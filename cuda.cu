#include "hip/hip_runtime.h"
%%cu

#include<stdio.h>

__global__ void Factorial(int *gpu_num,long int *gpu_res)
{
  int i;
  *gpu_res=1;
  for(i=1;i<=*gpu_num;i++)
  {
    *gpu_res = *gpu_res * i;      
  }
}

int main()
{
  int Number=4;  
  int *dev_number;
  long int *res, result;
  
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  
  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 
  Factorial<<<1,1>>>(dev_number,res);

  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);

  printf("\n\t Factorial of number %d is %ld \n",Number,result); 

  return 0; 
}
